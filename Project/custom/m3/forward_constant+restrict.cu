#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define TILE_WIDTH 16

__constant__ float MASK[7000];

__global__ void conv_forward_kernel(float* __restrict__ output, const float* __restrict__ input, const float* __restrict__ mask, const int B, const int M, const int C, const int H, const int W, const int K,const int S)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    output - output
    input - input
    mask - convolution kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    S - stride step length
    */

    const int H_out = (H - K)/S + 1;
    const int W_out = (W - K)/S + 1;
    // (void)H_out; // silence declared but never referenced warning. remove this line when you start working
    // (void)W_out; // silence declared but never referenced warning. remove this line when you start working

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = in_4d(0,0,0,0)
    // out_4d(0,0,0,0) = a

    #define out_4d(i3, i2, i1, i0) output[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
    #define in_4d(i3, i2, i1, i0) input[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
    #define mask_4d(i3, i2, i1, i0) MASK[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here
    int W_grid = ceil((float)((W - K)/S + 1)/TILE_WIDTH);
    int H_grid = ceil((float)((H - K)/S + 1)/TILE_WIDTH);
    int b = blockIdx.x;
    int m = blockIdx.y;
    int h = (blockIdx.z / W_grid) * TILE_WIDTH + threadIdx.y;
    int w = (blockIdx.z % W_grid) * TILE_WIDTH + threadIdx.x;
    float acc = 0.0f;

    if (K != 7){
        if (h < H_out && w < W_out){
            for (int c = 0; c < C; c++) {
                for (int p = 0; p < K; p++) {
                    for (int q = 0; q < K; q++) {
                        int in_h = h* S + p;
                        int in_w = w * S + q;
                        if (in_h < H && in_w < W) {
                            acc += in_4d(b, c, in_h, in_w) * mask_4d(m, c, p, q);
                        }
                    }
                }
            }
            out_4d(b, m, h, w) = acc;
        }
    }
    else{
        if (h < H_out && w < W_out){
            for (int c = 0; c < C; c++) {
                // unroll the loop
                acc += in_4d(b, c, h * S, w * S) * mask_4d(m, c, 0, 0);
                acc += in_4d(b, c, h * S, w * S + 1) * mask_4d(m, c, 0, 1);
                acc += in_4d(b, c, h * S, w * S + 2) * mask_4d(m, c, 0, 2);
                acc += in_4d(b, c, h * S, w * S + 3) * mask_4d(m, c, 0, 3);
                acc += in_4d(b, c, h * S, w * S + 4) * mask_4d(m, c, 0, 4);
                acc += in_4d(b, c, h * S, w * S + 5) * mask_4d(m, c, 0, 5);
                acc += in_4d(b, c, h * S, w * S + 6) * mask_4d(m, c, 0, 6);
                acc += in_4d(b, c, h * S + 1, w * S) * mask_4d(m, c, 1, 0);
                acc += in_4d(b, c, h * S + 1, w * S + 1) * mask_4d(m, c, 1, 1);
                acc += in_4d(b, c, h * S + 1, w * S + 2) * mask_4d(m, c, 1, 2);
                acc += in_4d(b, c, h * S + 1, w * S + 3) * mask_4d(m, c, 1, 3);
                acc += in_4d(b, c, h * S + 1, w * S + 4) * mask_4d(m, c, 1, 4);
                acc += in_4d(b, c, h * S + 1, w * S + 5) * mask_4d(m, c, 1, 5);
                acc += in_4d(b, c, h * S + 1, w * S + 6) * mask_4d(m, c, 1, 6);
                acc += in_4d(b, c, h * S + 2, w * S) * mask_4d(m, c, 2, 0);
                acc += in_4d(b, c, h * S + 2, w * S + 1) * mask_4d(m, c, 2, 1);
                acc += in_4d(b, c, h * S + 2, w * S + 2) * mask_4d(m, c, 2, 2);
                acc += in_4d(b, c, h * S + 2, w * S + 3) * mask_4d(m, c, 2, 3);
                acc += in_4d(b, c, h * S + 2, w * S + 4) * mask_4d(m, c, 2, 4);
                acc += in_4d(b, c, h * S + 2, w * S + 5) * mask_4d(m, c, 2, 5);
                acc += in_4d(b, c, h * S + 2, w * S + 6) * mask_4d(m, c, 2, 6);
                acc += in_4d(b, c, h * S + 3, w * S) * mask_4d(m, c, 3, 0);
                acc += in_4d(b, c, h * S + 3, w * S + 1) * mask_4d(m, c, 3, 1);
                acc += in_4d(b, c, h * S + 3, w * S + 2) * mask_4d(m, c, 3, 2);
                acc += in_4d(b, c, h * S + 3, w * S + 3) * mask_4d(m, c, 3, 3);
                acc += in_4d(b, c, h * S + 3, w * S + 4) * mask_4d(m, c, 3, 4);
                acc += in_4d(b, c, h * S + 3, w * S + 5) * mask_4d(m, c, 3, 5);
                acc += in_4d(b, c, h * S + 3, w * S + 6) * mask_4d(m, c, 3, 6);
                acc += in_4d(b, c, h * S + 4, w * S) * mask_4d(m, c, 4, 0);
                acc += in_4d(b, c, h * S + 4, w * S + 1) * mask_4d(m, c, 4, 1);
                acc += in_4d(b, c, h * S + 4, w * S + 2) * mask_4d(m, c, 4, 2);
                acc += in_4d(b, c, h * S + 4, w * S + 3) * mask_4d(m, c, 4, 3);
                acc += in_4d(b, c, h * S + 4, w * S + 4) * mask_4d(m, c, 4, 4);
                acc += in_4d(b, c, h * S + 4, w * S + 5) * mask_4d(m, c, 4, 5);
                acc += in_4d(b, c, h * S + 4, w * S + 6) * mask_4d(m, c, 4, 6);
                acc += in_4d(b, c, h * S + 5, w * S) * mask_4d(m, c, 5, 0);
                acc += in_4d(b, c, h * S + 5, w * S + 1) * mask_4d(m, c, 5, 1);
                acc += in_4d(b, c, h * S + 5, w * S + 2) * mask_4d(m, c, 5, 2);
                acc += in_4d(b, c, h * S + 5, w * S + 3) * mask_4d(m, c, 5, 3);
                acc += in_4d(b, c, h * S + 5, w * S + 4) * mask_4d(m, c, 5, 4);
                acc += in_4d(b, c, h * S + 5, w * S + 5) * mask_4d(m, c, 5, 5);
                acc += in_4d(b, c, h * S + 5, w * S + 6) * mask_4d(m, c, 5, 6);
                acc += in_4d(b, c, h * S + 6, w * S) * mask_4d(m, c, 6, 0);
                acc += in_4d(b, c, h * S + 6, w * S + 1) * mask_4d(m, c, 6, 1);
                acc += in_4d(b, c, h * S + 6, w * S + 2) * mask_4d(m, c, 6, 2);
                acc += in_4d(b, c, h * S + 6, w * S + 3) * mask_4d(m, c, 6, 3);
                acc += in_4d(b, c, h * S + 6, w * S + 4) * mask_4d(m, c, 6, 4);
                acc += in_4d(b, c, h * S + 6, w * S + 5) * mask_4d(m, c, 6, 5);
                acc += in_4d(b, c, h * S + 6, w * S + 6) * mask_4d(m, c, 6, 6);
            }
        }
    }



    #undef out_4d
    #undef in_4d
    #undef mask_4d
}

	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_output, const float *host_input, const float *host_mask, float **device_output_ptr, float **device_input_ptr, float **device_mask_ptr, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }

    hipMalloc((void**)device_output_ptr, B * M * ((H - K)/S + 1) * ((W - K)/S + 1) * sizeof(float));
    hipMalloc((void**)device_input_ptr, B * C * H * W * sizeof(float));
    hipMalloc((void**)device_mask_ptr, M * C * K * K * sizeof(float));
    hipMemcpy(*device_input_ptr, host_input, B * C * H * W * sizeof(float), hipMemcpyHostToDevice);
    // hipMemcpy(*device_mask_ptr, host_mask, M * C * K * K * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(MASK), host_mask, M * C * K * K * sizeof(float));

   
}


__host__ void GPUInterface::conv_forward_gpu(float *device_output, const float *device_input, const float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Set the kernel dimensions and call the kernel
    int W_grid = ceil((float)((W - K)/S + 1)/TILE_WIDTH);
    int H_grid = ceil((float)((H - K)/S + 1)/TILE_WIDTH);
    int Y = W_grid * H_grid;
    dim3 dimgrid(B, M, Y);
    dim3 dimblock(TILE_WIDTH, TILE_WIDTH, 1);
    conv_forward_kernel<<<dimgrid, dimblock>>>(device_output, device_input, device_mask, B, M, C, H, W, K, S);

}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_output, float *device_output, float *device_input, float *device_mask, const int B, const int M, const int C, const int H, const int W, const int K, const int S)
{
    // Copy the output back to host
    hipMemcpy(host_output, device_output, B * M * ((W - K)/S + 1) * ((W - K)/S + 1) * sizeof(float), hipMemcpyDeviceToHost);
   
    // Free device memory
    hipFree(device_output);
    hipFree(device_input);

}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
